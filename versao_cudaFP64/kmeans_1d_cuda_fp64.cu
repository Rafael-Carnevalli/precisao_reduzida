/* kmeans_1d_cuda_fp64.cu
   K-means 1D (CUDA C++), implementação para GPU com FP64.
   - Usa double para precisão de 64 bits e kernels CUDA para paralelismo.
   - Lê X e C_init de CSVs, executa na GPU e salva os resultados.

   Compilar com o CUDA Toolkit:
   nvcc -O2 -arch=sm_89 kmeans_1d_cuda_fp64.cu -o kmeans_1d_cuda_fp64
   (-arch=sm_89 é para a RTX 4060, Compute Capability 8.9)

   Uso:      ./kmeans_1d_cuda_fp64 dados.csv centroides_iniciais.csv [max_iter=50] [eps=1e-4] [assign.csv] [centroids.csv]
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <float.h>

// Macro para checagem de erros CUDA
#define CUDA_CHECK(err) { \
    if (err != hipSuccess) { \
        fprintf(stderr, "Erro CUDA em %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(1); \
    } \
}

/* ---------- Funções de I/O (executam na CPU) ---------- */
static int count_rows(const char *path){
    FILE *f = fopen(path, "r");
    if(!f){ fprintf(stderr,"Erro ao abrir %s\n", path); exit(1); }
    int rows=0; char line[8192];
    while(fgets(line,sizeof(line),f)){
        int only_ws=1;
        for(char *p=line; *p; p++){
            if(*p!=' ' && *p!='\t' && *p!='\n' && *p!='\r'){ only_ws=0; break; }
        }
        if(!only_ws) rows++;
    }
    fclose(f);
    return rows;
}

static double *read_csv_1col(const char *path, int *n_out){
    int R = count_rows(path);
    if(R<=0){ fprintf(stderr,"Arquivo vazio: %s\n", path); exit(1); }
    double *A = (double*)malloc((size_t)R * sizeof(double));
    if(!A){ fprintf(stderr,"Sem memoria para %d linhas\n", R); exit(1); }
    FILE *f = fopen(path, "r");
    if(!f){ fprintf(stderr,"Erro ao abrir %s\n", path); free(A); exit(1); }
    char line[8192];
    int r=0;
    while(fgets(line,sizeof(line),f)){
        int only_ws=1;
        for(char *p=line; *p; p++){
            if(*p!=' ' && *p!='\t' && *p!='\n' && *p!='\r'){ only_ws=0; break; }
        }
        if(only_ws) continue;
        const char *delim = ",; \t";
        char *tok = strtok(line, delim);
        if(!tok){ fprintf(stderr,"Linha %d sem valor em %s\n", r+1, path); free(A); fclose(f); exit(1); }
        A[r] = atof(tok);
        r++;
        if(r>R) break;
    }
    fclose(f);
    *n_out = R;
    return A;
}

static void write_assign_csv(const char *path, const int *assign, int N){
    if(!path) return;
    FILE *f = fopen(path, "w");
    if(!f){ fprintf(stderr,"Erro ao abrir %s para escrita\n", path); return; }
    for(int i=0;i<N;i++) fprintf(f, "%d\n", assign[i]);
    fclose(f);
}

static void write_centroids_csv(const char *path, const double *C, int K){
    if(!path) return;
    FILE *f = fopen(path, "w");
    if(!f){ fprintf(stderr,"Erro ao abrir %s para escrita\n", path); return; }
    for(int c=0;c<K;c++) {
        fprintf(f, "%.6f\n", C[c]);
    }
    fclose(f);
}

/* ---------- Kernels CUDA (executam na GPU) ---------- */

__global__ void assignment_kernel(const double *X, const double *C, int *assign, double *sse_sum, int N, int K) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    int best = -1;
    double bestd = DBL_MAX;
    for (int c = 0; c < K; c++) {
        double diff = X[i] - C[c];
        double d = diff * diff;
        if (d < bestd) {
            bestd = d;
            best = c;
        }
    }
    assign[i] = best;
    atomicAdd(sse_sum, bestd);
}

__global__ void zero_sums_kernel(double *sum, int *cnt, int K) {
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    if (c >= K) return;
    sum[c] = 0.0;
    cnt[c] = 0;
}

__global__ void update_sums_kernel(const double *X, const int *assign, double *sum, int *cnt, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    int a = assign[i];
    atomicAdd(&sum[a], X[i]);
    atomicAdd(&cnt[a], 1);
}

__global__ void update_centroids_kernel(const double *X, double *C, const double *sum, const int *cnt, int K) {
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    if (c >= K) return;
    if (cnt[c] > 0) {
        C[c] = sum[c] / (double)cnt[c];
    } else {
        C[c] = X[0]; // Estratégia naive para cluster vazio
    }
}

/* ---------- Orquestrador (executa na CPU) ---------- */
static void kmeans_1d(const double *h_X, double *h_C, int *h_assign,
                      int N, int K, int max_iter, double eps,
                      int *iters_out, double *sse_out)
{
    // Alocação de memória na GPU
    double *d_X, *d_C, *d_sum;
    int *d_assign, *d_cnt;
    double *d_sse_sum;
    CUDA_CHECK(hipMalloc(&d_X, N * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_C, K * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_assign, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_sum, K * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_cnt, K * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_sse_sum, sizeof(double)));

    // Transferência de dados da CPU para a GPU
    CUDA_CHECK(hipMemcpy(d_X, h_X, N * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_C, h_C, K * sizeof(double), hipMemcpyHostToDevice));

    int threads_per_block = 256;
    int blocks_N = (N + threads_per_block - 1) / threads_per_block;
    int blocks_K = (K + threads_per_block - 1) / threads_per_block;

    double sse = 0.0, prev_sse = DBL_MAX;
    int it;
    for (it = 0; it < max_iter; it++) {
        // 1. Passo de Atribuição
        CUDA_CHECK(hipMemset(d_sse_sum, 0, sizeof(double)));
        assignment_kernel<<<blocks_N, threads_per_block>>>(d_X, d_C, d_assign, d_sse_sum, N, K);
        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipMemcpy(&sse, d_sse_sum, sizeof(double), hipMemcpyDeviceToHost));

        // 2. Checagem de convergência (na CPU)
        double rel = fabs(sse - prev_sse) / (prev_sse > 0.0 ? prev_sse : 1.0);
        if (rel < eps) { it++; break; }
        prev_sse = sse;

        // 3. Passo de Atualização
        zero_sums_kernel<<<blocks_K, threads_per_block>>>(d_sum, d_cnt, K);
        update_sums_kernel<<<blocks_N, threads_per_block>>>(d_X, d_assign, d_sum, d_cnt, N);
        update_centroids_kernel<<<blocks_K, threads_per_block>>>(d_X, d_C, d_sum, d_cnt, K);
        CUDA_CHECK(hipDeviceSynchronize());
    }

    // Copia os resultados de volta para a CPU
    CUDA_CHECK(hipMemcpy(h_C, d_C, K * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_assign, d_assign, N * sizeof(int), hipMemcpyDeviceToHost));

    // Libera memória da GPU
    CUDA_CHECK(hipFree(d_X));
    CUDA_CHECK(hipFree(d_C));
    CUDA_CHECK(hipFree(d_assign));
    CUDA_CHECK(hipFree(d_sum));
    CUDA_CHECK(hipFree(d_cnt));
    CUDA_CHECK(hipFree(d_sse_sum));

    *iters_out = it;
    *sse_out = sse;
}

/* ---------- main (executa na CPU) ---------- */
int main(int argc, char **argv){
    if(argc < 3){
        printf("Uso: %s dados.csv centroides_iniciais.csv [max_iter=50] [eps=1e-4] [assign.csv] [centroids.csv]\n", argv[0]);
        return 1;
    }
    const char *pathX = argv[1];
    const char *pathC = argv[2];
    int max_iter = (argc>3)? atoi(argv[3]) : 50;
    double eps   = (argc>4)? atof(argv[4]) : 1e-4;
    const char *outAssign   = (argc>5)? argv[5] : NULL;
    const char *outCentroid = (argc>6)? argv[6] : NULL;

    if(max_iter <= 0 || eps <= 0.0){
        fprintf(stderr,"Parâmetros inválidos: max_iter>0 e eps>0\n");
        return 1;
    }

    int N=0, K=0;
    double *X = read_csv_1col(pathX, &N);
    double *C = read_csv_1col(pathC, &K);
    int *assign = (int*)malloc((size_t)N * sizeof(int));
    if(!assign){ fprintf(stderr,"Sem memoria para assign\n"); free(X); free(C); return 1; }

    clock_t t0 = clock();
    int iters = 0; double sse = 0.0;
    kmeans_1d(X, C, assign, N, K, max_iter, eps, &iters, &sse);
    clock_t t1 = clock();
    double ms = 1000.0 * (double)(t1 - t0) / (double)CLOCKS_PER_SEC;

    printf("K-means 1D (CUDA FP64)\n");
    printf("N=%d K=%d max_iter=%d eps=%g\n", N, K, max_iter, eps);
    printf("Iterações: %d | SSE final: %.6f | Tempo: %.1f ms\n", iters, sse, ms);

    write_assign_csv(outAssign, assign, N);
    write_centroids_csv(outCentroid, C, K);

    free(assign); free(X); free(C);
    return 0;
}
