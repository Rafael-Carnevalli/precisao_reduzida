/* kmeans_1d_cuda_fp32.cu
   K-means 1D (CUDA C++), implementação para GPU com FP32.
   - Usa float para precisão de 32 bits e kernels CUDA para paralelismo.
   - Lê X e C_init de CSVs, executa na GPU e salva os resultados.

   Compilar com o CUDA Toolkit:
   nvcc -O2 -arch=sm_89 kmeans_1d_cuda_fp32.cu -o kmeans_1d_cuda_fp32
   (-arch=sm_89 é para a RTX 4060, Compute Capability 8.9)

   Uso:      ./kmeans_1d_cuda_fp32 dados.csv centroides_iniciais.csv [max_iter=50] [eps=1e-4] [assign.csv] [centroids.csv]
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <float.h>

// Macro para checagem de erros CUDA
#define CUDA_CHECK(err) { \
    if (err != hipSuccess) { \
        fprintf(stderr, "Erro CUDA em %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(1); \
    } \
}

/* ---------- Funções de I/O (executam na CPU) ---------- */
static int count_rows(const char *path){
    FILE *f = fopen(path, "r");
    if(!f){ fprintf(stderr,"Erro ao abrir %s\n", path); exit(1); }
    int rows=0; char line[8192];
    while(fgets(line,sizeof(line),f)){
        int only_ws=1;
        for(char *p=line; *p; p++){
            if(*p!=' ' && *p!='\t' && *p!='\n' && *p!='\r'){ only_ws=0; break; }
        }
        if(!only_ws) rows++;
    }
    fclose(f);
    return rows;
}

static float *read_csv_1col(const char *path, int *n_out){
    int R = count_rows(path);
    if(R<=0){ fprintf(stderr,"Arquivo vazio: %s\n", path); exit(1); }
    float *A = (float*)malloc((size_t)R * sizeof(float));
    if(!A){ fprintf(stderr,"Sem memoria para %d linhas\n", R); exit(1); }
    FILE *f = fopen(path, "r");
    if(!f){ fprintf(stderr,"Erro ao abrir %s\n", path); free(A); exit(1); }
    char line[8192];
    int r=0;
    while(fgets(line,sizeof(line),f)){
        int only_ws=1;
        for(char *p=line; *p; p++){
            if(*p!=' ' && *p!='\t' && *p!='\n' && *p!='\r'){ only_ws=0; break; }
        }
        if(only_ws) continue;
        const char *delim = ",; \t";
        char *tok = strtok(line, delim);
        if(!tok){ fprintf(stderr,"Linha %d sem valor em %s\n", r+1, path); free(A); fclose(f); exit(1); }
        A[r] = strtof(tok, NULL);
        r++;
        if(r>R) break;
    }
    fclose(f);
    *n_out = R;
    return A;
}

static void write_assign_csv(const char *path, const int *assign, int N){
    if(!path) return;
    FILE *f = fopen(path, "w");
    if(!f){ fprintf(stderr,"Erro ao abrir %s para escrita\n", path); return; }
    for(int i=0;i<N;i++) fprintf(f, "%d\n", assign[i]);
    fclose(f);
}

static void write_centroids_csv(const char *path, const float *C, int K){
    if(!path) return;
    FILE *f = fopen(path, "w");
    if(!f){ fprintf(stderr,"Erro ao abrir %s para escrita\n", path); return; }
    for(int c=0;c<K;c++) {
        fprintf(f, "%.6f\n", C[c]);
    }
    fclose(f);
}

/* ---------- Kernels CUDA (executam na GPU) ---------- */

__global__ void assignment_kernel(const float *X, const float *C, int *assign, float *sse_sum, int N, int K) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    int best = -1;
    float bestd = FLT_MAX;
    for (int c = 0; c < K; c++) {
        float diff = X[i] - C[c];
        float d = diff * diff;
        if (d < bestd) {
            bestd = d;
            best = c;
        }
    }
    assign[i] = best;
    atomicAdd(sse_sum, bestd);
}

__global__ void zero_sums_kernel(float *sum, int *cnt, int K) {
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    if (c >= K) return;
    sum[c] = 0.0f;
    cnt[c] = 0;
}

__global__ void update_sums_kernel(const float *X, const int *assign, float *sum, int *cnt, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    int a = assign[i];
    atomicAdd(&sum[a], X[i]);
    atomicAdd(&cnt[a], 1);
}

__global__ void update_centroids_kernel(const float *X, float *C, const float *sum, const int *cnt, int K) {
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    if (c >= K) return;
    if (cnt[c] > 0) {
        C[c] = sum[c] / (float)cnt[c];
    } else {
        C[c] = X[0]; // Estratégia naive para cluster vazio
    }
}

/* ---------- Orquestrador (executa na CPU) ---------- */
static void kmeans_1d(const float *h_X, float *h_C, int *h_assign,
                      int N, int K, int max_iter, float eps,
                      int *iters_out, float *sse_out)
{
    // Alocação de memória na GPU
    float *d_X, *d_C, *d_sum;
    int *d_assign, *d_cnt;
    float *d_sse_sum;
    CUDA_CHECK(hipMalloc(&d_X, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_C, K * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_assign, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_sum, K * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_cnt, K * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_sse_sum, sizeof(float)));

    // Transferência de dados da CPU para a GPU
    CUDA_CHECK(hipMemcpy(d_X, h_X, N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_C, h_C, K * sizeof(float), hipMemcpyHostToDevice));

    int threads_per_block = 256;
    int blocks_N = (N + threads_per_block - 1) / threads_per_block;
    int blocks_K = (K + threads_per_block - 1) / threads_per_block;

    float sse = 0.0f, prev_sse = FLT_MAX;
    int it;
    for (it = 0; it < max_iter; it++) {
        // 1. Passo de Atribuição
        CUDA_CHECK(hipMemset(d_sse_sum, 0, sizeof(float)));
        assignment_kernel<<<blocks_N, threads_per_block>>>(d_X, d_C, d_assign, d_sse_sum, N, K);
        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipMemcpy(&sse, d_sse_sum, sizeof(float), hipMemcpyDeviceToHost));

        // 2. Checagem de convergência (na CPU)
        float rel = fabsf(sse - prev_sse) / (prev_sse > 0.0f ? prev_sse : 1.0f);
        if (rel < eps) { it++; break; }
        prev_sse = sse;

        // 3. Passo de Atualização
        zero_sums_kernel<<<blocks_K, threads_per_block>>>(d_sum, d_cnt, K);
        update_sums_kernel<<<blocks_N, threads_per_block>>>(d_X, d_assign, d_sum, d_cnt, N);
        update_centroids_kernel<<<blocks_K, threads_per_block>>>(d_X, d_C, d_sum, d_cnt, K);
        CUDA_CHECK(hipDeviceSynchronize());
    }

    // Copia os resultados de volta para a CPU
    CUDA_CHECK(hipMemcpy(h_C, d_C, K * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_assign, d_assign, N * sizeof(int), hipMemcpyDeviceToHost));

    // Libera memória da GPU
    CUDA_CHECK(hipFree(d_X));
    CUDA_CHECK(hipFree(d_C));
    CUDA_CHECK(hipFree(d_assign));
    CUDA_CHECK(hipFree(d_sum));
    CUDA_CHECK(hipFree(d_cnt));
    CUDA_CHECK(hipFree(d_sse_sum));

    *iters_out = it;
    *sse_out = sse;
}

/* ---------- main (executa na CPU) ---------- */
int main(int argc, char **argv){
    if(argc < 3){
        printf("Uso: %s dados.csv centroides_iniciais.csv [max_iter=50] [eps=1e-4] [assign.csv] [centroids.csv]\n", argv[0]);
        return 1;
    }
    const char *pathX = argv[1];
    const char *pathC = argv[2];
    int max_iter = (argc>3)? atoi(argv[3]) : 50;
    float eps   = (argc>4)? strtof(argv[4], NULL) : 1e-4f;
    const char *outAssign   = (argc>5)? argv[5] : NULL;
    const char *outCentroid = (argc>6)? argv[6] : NULL;

    if(max_iter <= 0 || eps <= 0.0f){
        fprintf(stderr,"Parâmetros inválidos: max_iter>0 e eps>0\n");
        return 1;
    }

    int N=0, K=0;
    float *X = read_csv_1col(pathX, &N);
    float *C = read_csv_1col(pathC, &K);
    int *assign = (int*)malloc((size_t)N * sizeof(int));
    if(!assign){ fprintf(stderr,"Sem memoria para assign\n"); free(X); free(C); return 1; }

    clock_t t0 = clock();
    int iters = 0; float sse = 0.0f;
    kmeans_1d(X, C, assign, N, K, max_iter, eps, &iters, &sse);
    clock_t t1 = clock();
    double ms = 1000.0 * (double)(t1 - t0) / (double)CLOCKS_PER_SEC;

    printf("K-means 1D (CUDA FP32)\n");
    printf("N=%d K=%d max_iter=%d eps=%g\n", N, K, max_iter, eps);
    printf("Iterações: %d | SSE final: %.6f | Tempo: %.1f ms\n", iters, sse, ms);

    write_assign_csv(outAssign, assign, N);
    write_centroids_csv(outCentroid, C, K);

    free(assign); free(X); free(C);
    return 0;
}
